#include "hip/hip_runtime.h"
#include "infer.hpp"
#include "yolo.hpp"

namespace yolo {

using namespace std;

#define GPU_BLOCK_THREADS 512
#define checkRuntime(call)                                                                                             \
    do {                                                                                                               \
        auto ___call__ret_code__ = (call);                                                                             \
        if (___call__ret_code__ != hipSuccess) {                                                                      \
            INFO("CUDA Runtime error💥 %s # %s, code = %s [ %d ]", #call, hipGetErrorString(___call__ret_code__),      \
                 hipGetErrorName(___call__ret_code__), ___call__ret_code__);                                          \
            abort();                                                                                                   \
        }                                                                                                              \
    } while (0)

#define checkKernel(...)                                                                                               \
    do {                                                                                                               \
        { (__VA_ARGS__); }                                                                                             \
        checkRuntime(hipPeekAtLastError());                                                                           \
    } while (0)

enum class NormType : int { None = 0, MeanStd = 1, AlphaBeta = 2 };

enum class ChannelType : int { None = 0, SwapRB = 1 };

/* 归一化操作，可以支持均值标准差，alpha beta，和swap RB */
struct Norm {
    float mean[3];
    float std[3];
    float alpha, beta;
    NormType type            = NormType::None;
    ChannelType channel_type = ChannelType::None;

    // out = (x * alpha - mean) / std
    static Norm mean_std(const float mean[3], const float std[3], float alpha = 1 / 255.0f,
                         ChannelType channel_type = ChannelType::None);

    // out = x * alpha + beta
    static Norm alpha_beta(float alpha, float beta = 0, ChannelType channel_type = ChannelType::None);

    // None
    static Norm None();
};

Norm Norm::mean_std(const float mean[3], const float std[3], float alpha, ChannelType channel_type) {
    Norm out;
    out.type         = NormType::MeanStd;
    out.alpha        = alpha;
    out.channel_type = channel_type;
    memcpy(out.mean, mean, sizeof(out.mean));
    memcpy(out.std, std, sizeof(out.std));
    return out;
}

Norm Norm::alpha_beta(float alpha, float beta, ChannelType channel_type) {
    Norm out;
    out.type         = NormType::AlphaBeta;
    out.alpha        = alpha;
    out.beta         = beta;
    out.channel_type = channel_type;
    return out;
}

Norm Norm::None() {
    return Norm();
}

const int NUM_BOX_ELEMENT = 7;  // left, top, right, bottom, confidence, class, keepflag
const int MAX_IMAGE_BOXES = 1024;
inline int upbound(int n, int align = 32) {
    return (n + align - 1) / align * align;
}
static __device__ void affine_project(float* matrix, float x, float y, float* ox, float* oy) {
    *ox = matrix[0] * x + matrix[1] * y + matrix[2];
    *oy = matrix[3] * x + matrix[4] * y + matrix[5];
}

static __global__ void decode_kernel(float* predict, int num_bboxes, int num_classes, float confidence_threshold,
                                     float* invert_affine_matrix, float* parray, int MAX_IMAGE_BOXES) {
    int position = blockDim.x * blockIdx.x + threadIdx.x;
    if (position >= num_bboxes)
        return;

    float* pitem     = predict + (5 + num_classes) * position;
    float objectness = pitem[4];
    if (objectness < confidence_threshold)
        return;

    float* class_confidence = pitem + 5;
    float confidence        = *class_confidence++;
    int label               = 0;
    for (int i = 1; i < num_classes; ++i, ++class_confidence) {
        if (*class_confidence > confidence) {
            confidence = *class_confidence;
            label      = i;
        }
    }

    confidence *= objectness;
    if (confidence < confidence_threshold)
        return;

    int index = atomicAdd(parray, 1);
    if (index >= MAX_IMAGE_BOXES)
        return;

    float cx     = *pitem++;
    float cy     = *pitem++;
    float width  = *pitem++;
    float height = *pitem++;
    float left   = cx - width * 0.5f;
    float top    = cy - height * 0.5f;
    float right  = cx + width * 0.5f;
    float bottom = cy + height * 0.5f;
    affine_project(invert_affine_matrix, left, top, &left, &top);
    affine_project(invert_affine_matrix, right, bottom, &right, &bottom);

    float* pout_item = parray + 1 + index * NUM_BOX_ELEMENT;
    *pout_item++     = left;
    *pout_item++     = top;
    *pout_item++     = right;
    *pout_item++     = bottom;
    *pout_item++     = confidence;
    *pout_item++     = label;
    *pout_item++     = 1;  // 1 = keep, 0 = ignore
}

static __device__ float box_iou(float aleft, float atop, float aright, float abottom, float bleft, float btop,
                                float bright, float bbottom) {
    float cleft   = max(aleft, bleft);
    float ctop    = max(atop, btop);
    float cright  = min(aright, bright);
    float cbottom = min(abottom, bbottom);

    float c_area = max(cright - cleft, 0.0f) * max(cbottom - ctop, 0.0f);
    if (c_area == 0.0f)
        return 0.0f;

    float a_area = max(0.0f, aright - aleft) * max(0.0f, abottom - atop);
    float b_area = max(0.0f, bright - bleft) * max(0.0f, bbottom - btop);
    return c_area / (a_area + b_area - c_area);
}

static __global__ void fast_nms_kernel(float* bboxes, int MAX_IMAGE_BOXES, float threshold) {
    int position = (blockDim.x * blockIdx.x + threadIdx.x);
    int count    = min((int)*bboxes, MAX_IMAGE_BOXES);
    if (position >= count)
        return;

    // left, top, right, bottom, confidence, class, keepflag
    float* pcurrent = bboxes + 1 + position * NUM_BOX_ELEMENT;
    for (int i = 0; i < count; ++i) {
        float* pitem = bboxes + 1 + i * NUM_BOX_ELEMENT;
        if (i == position || pcurrent[5] != pitem[5])
            continue;

        if (pitem[4] >= pcurrent[4]) {
            if (pitem[4] == pcurrent[4] && i < position)
                continue;

            float iou =
                box_iou(pcurrent[0], pcurrent[1], pcurrent[2], pcurrent[3], pitem[0], pitem[1], pitem[2], pitem[3]);

            if (iou > threshold) {
                pcurrent[6] = 0;  // 1=keep, 0=ignore
                return;
            }
        }
    }
}

static dim3 grid_dims(int numJobs) {
    int numBlockThreads = numJobs < GPU_BLOCK_THREADS ? numJobs : GPU_BLOCK_THREADS;
    return dim3(((numJobs + numBlockThreads - 1) / (float)numBlockThreads));
}

static dim3 block_dims(int numJobs) {
    return numJobs < GPU_BLOCK_THREADS ? numJobs : GPU_BLOCK_THREADS;
}

static void decode_kernel_invoker(float* predict, int num_bboxes, int num_classes, float confidence_threshold,
                                  float nms_threshold, float* invert_affine_matrix, float* parray, int MAX_IMAGE_BOXES,
                                  hipStream_t stream) {
    auto grid  = grid_dims(num_bboxes);
    auto block = block_dims(num_bboxes);

    /* 如果核函数有波浪线，没关系，他是正常的，你只是看不顺眼罢了 */
    checkKernel(decode_kernel<<<grid, block, 0, stream>>>(predict, num_bboxes, num_classes, confidence_threshold,
                                                          invert_affine_matrix, parray, MAX_IMAGE_BOXES));

    grid  = grid_dims(MAX_IMAGE_BOXES);
    block = block_dims(MAX_IMAGE_BOXES);
    checkKernel(fast_nms_kernel<<<grid, block, 0, stream>>>(parray, MAX_IMAGE_BOXES, nms_threshold));
}

static __global__ void warp_affine_bilinear_and_normalize_plane_kernel(uint8_t* src, int src_line_size, int src_width,
                                                                       int src_height, float* dst, int dst_width,
                                                                       int dst_height, uint8_t const_value_st,
                                                                       float* warp_affine_matrix_2_3, Norm norm) {
    int dx = blockDim.x * blockIdx.x + threadIdx.x;
    int dy = blockDim.y * blockIdx.y + threadIdx.y;
    if (dx >= dst_width || dy >= dst_height)
        return;

    float m_x1 = warp_affine_matrix_2_3[0];
    float m_y1 = warp_affine_matrix_2_3[1];
    float m_z1 = warp_affine_matrix_2_3[2];
    float m_x2 = warp_affine_matrix_2_3[3];
    float m_y2 = warp_affine_matrix_2_3[4];
    float m_z2 = warp_affine_matrix_2_3[5];

    float src_x = m_x1 * dx + m_y1 * dy + m_z1;
    float src_y = m_x2 * dx + m_y2 * dy + m_z2;
    float c0, c1, c2;

    if (src_x <= -1 || src_x >= src_width || src_y <= -1 || src_y >= src_height) {
        // out of range
        c0 = const_value_st;
        c1 = const_value_st;
        c2 = const_value_st;
    } else {
        int y_low  = floorf(src_y);
        int x_low  = floorf(src_x);
        int y_high = y_low + 1;
        int x_high = x_low + 1;

        uint8_t const_value[] = {const_value_st, const_value_st, const_value_st};
        float ly              = src_y - y_low;
        float lx              = src_x - x_low;
        float hy              = 1 - ly;
        float hx              = 1 - lx;
        float w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;
        uint8_t* v1 = const_value;
        uint8_t* v2 = const_value;
        uint8_t* v3 = const_value;
        uint8_t* v4 = const_value;
        if (y_low >= 0) {
            if (x_low >= 0)
                v1 = src + y_low * src_line_size + x_low * 3;

            if (x_high < src_width)
                v2 = src + y_low * src_line_size + x_high * 3;
        }

        if (y_high < src_height) {
            if (x_low >= 0)
                v3 = src + y_high * src_line_size + x_low * 3;

            if (x_high < src_width)
                v4 = src + y_high * src_line_size + x_high * 3;
        }

        // same to opencv
        c0 = floorf(w1 * v1[0] + w2 * v2[0] + w3 * v3[0] + w4 * v4[0] + 0.5f);
        c1 = floorf(w1 * v1[1] + w2 * v2[1] + w3 * v3[1] + w4 * v4[1] + 0.5f);
        c2 = floorf(w1 * v1[2] + w2 * v2[2] + w3 * v3[2] + w4 * v4[2] + 0.5f);
    }

    if (norm.channel_type == ChannelType::SwapRB) {
        float t = c2;
        c2      = c0;
        c0      = t;
    }

    if (norm.type == NormType::MeanStd) {
        c0 = (c0 * norm.alpha - norm.mean[0]) / norm.std[0];
        c1 = (c1 * norm.alpha - norm.mean[1]) / norm.std[1];
        c2 = (c2 * norm.alpha - norm.mean[2]) / norm.std[2];
    } else if (norm.type == NormType::AlphaBeta) {
        c0 = c0 * norm.alpha + norm.beta;
        c1 = c1 * norm.alpha + norm.beta;
        c2 = c2 * norm.alpha + norm.beta;
    }

    int area       = dst_width * dst_height;
    float* pdst_c0 = dst + dy * dst_width + dx;
    float* pdst_c1 = pdst_c0 + area;
    float* pdst_c2 = pdst_c1 + area;
    *pdst_c0       = c0;
    *pdst_c1       = c1;
    *pdst_c2       = c2;
}

static void warp_affine_bilinear_and_normalize_plane(uint8_t* src, int src_line_size, int src_width, int src_height,
                                                     float* dst, int dst_width, int dst_height, float* matrix_2_3,
                                                     uint8_t const_value, const Norm& norm, hipStream_t stream) {
    dim3 grid((dst_width + 31) / 32, (dst_height + 31) / 32);
    dim3 block(32, 32);

    checkKernel(warp_affine_bilinear_and_normalize_plane_kernel<<<grid, block, 0, stream>>>(
        src, src_line_size, src_width, src_height, dst, dst_width, dst_height, const_value, matrix_2_3, norm));
}

const char* type_name(Type type) {
    switch (type) {
        case Type::V5:
            return "YoloV5";
        case Type::V3:
            return "YoloV3";
        case Type::V7:
            return "YoloV7";
        case Type::X:
            return "YoloX";
        default:
            return "Unknow";
    }
}

struct AffineMatrix {
    float i2d[6];  // image to dst(network), 2x3 matrix
    float d2i[6];  // dst to image, 2x3 matrix

    void compute(const std::tuple<int, int>& from, const std::tuple<int, int>& to) {
        float scale_x = get<0>(to) / (float)get<0>(from);
        float scale_y = get<1>(to) / (float)get<1>(from);
        float scale   = std::min(scale_x, scale_y);
        i2d[0]        = scale;
        i2d[1]        = 0;
        i2d[2]        = -scale * get<0>(from) * 0.5 + get<0>(to) * 0.5 + scale * 0.5 - 0.5;
        i2d[3]        = 0;
        i2d[4]        = scale;
        i2d[5]        = -scale * get<1>(from) * 0.5 + get<1>(to) * 0.5 + scale * 0.5 - 0.5;

        double D   = i2d[0] * i2d[4] - i2d[1] * i2d[3];
        D          = D != 0. ? double(1.) / D : double(0.);
        double A11 = i2d[4] * D, A22 = i2d[0] * D, A12 = -i2d[1] * D, A21 = -i2d[3] * D;
        double b1 = -A11 * i2d[2] - A12 * i2d[5];
        double b2 = -A21 * i2d[2] - A22 * i2d[5];

        d2i[0] = A11;
        d2i[1] = A12;
        d2i[2] = b1;
        d2i[3] = A21;
        d2i[4] = A22;
        d2i[5] = b2;
    }
};

class InferImpl : public Infer {
public:
    shared_ptr<trt::Infer> trt_;
    string engine_file_;
    Type type_;
    float confidence_threshold_;
    float nms_threshold_;
    vector<shared_ptr<trt::Memory<unsigned char>>> preprocess_buffers_;
    trt::Memory<float> input_buffer_, predict_buffer_, output_boxarray_;
    int input_width_, input_height_;
    Norm normalize_;
    vector<int> network_output_dims_;
    int num_classes_ = 0;

    void adjust_memory() {
        // the inference batch_size
        int batch_size     = trt_->run_dims(0)[0];
        size_t input_numel = input_width_ * input_height_ * 3;
        input_buffer_.gpu(batch_size * input_numel);
        predict_buffer_.gpu(batch_size * network_output_dims_[1] * network_output_dims_[2]);
        output_boxarray_.gpu(batch_size * (32 + MAX_IMAGE_BOXES * NUM_BOX_ELEMENT));
        output_boxarray_.cpu(batch_size * (32 + MAX_IMAGE_BOXES * NUM_BOX_ELEMENT));

        if (preprocess_buffers_.size() < batch_size) {
            for (int i = preprocess_buffers_.size(); i < batch_size; ++i)
                preprocess_buffers_.push_back(make_shared<trt::Memory<unsigned char>>());
        }
    }

    void preprocess(int ibatch, const Image& image, shared_ptr<trt::Memory<unsigned char>> preprocess_buffer,
                    void* stream = nullptr) {
        AffineMatrix affine;
        affine.compute(make_tuple(image.width, image.height), make_tuple(input_width_, input_height_));

        size_t input_numel          = input_width_ * input_height_ * 3;
        float* input_device         = input_buffer_.gpu() + ibatch * input_numel;
        size_t size_image           = image.width * image.height * 3;
        size_t size_matrix          = upbound(sizeof(affine.d2i), 32);
        uint8_t* gpu_workspace      = preprocess_buffer->gpu(size_matrix + size_image);
        float* affine_matrix_device = (float*)gpu_workspace;
        uint8_t* image_device       = gpu_workspace + size_matrix;

        uint8_t* cpu_workspace    = preprocess_buffer->cpu(size_matrix + size_image);
        float* affine_matrix_host = (float*)cpu_workspace;
        uint8_t* image_host       = cpu_workspace + size_matrix;

        // speed up
        hipStream_t stream_ = (hipStream_t)stream;
        if (image.use_device) {
            checkRuntime(hipMemcpyAsync(image_device, image.bgrptr, size_image, hipMemcpyDeviceToDevice, stream_));
        } else {
            memcpy(image_host, image.bgrptr, size_image);
	    //checkRuntime(hipMemcpyAsync(image_device, image.bgrptr, size_image, hipMemcpyHostToDevice, stream_));

            checkRuntime(hipMemcpyAsync(image_device, image_host, size_image, hipMemcpyHostToDevice, stream_));
        }
        memcpy(affine_matrix_host, affine.d2i, sizeof(affine.d2i));
        checkRuntime(hipMemcpyAsync(affine_matrix_device, affine_matrix_host, sizeof(affine.d2i),
                                     hipMemcpyHostToDevice, stream_));

        warp_affine_bilinear_and_normalize_plane(image_device, image.width * 3, image.width, image.height, input_device,
                                                 input_width_, input_height_, affine_matrix_device, 114, normalize_,
                                                 stream_);
    }

    bool load(const string& engine_file, Type type, float confidence_threshold, float nms_threshold) {
        trt_ = trt::load(engine_file);
        if (trt_ == nullptr)
            return false;

        this->type_                 = type;
        this->confidence_threshold_ = confidence_threshold;
        this->nms_threshold_        = nms_threshold;

        if (type == Type::V5 || type == Type::V3 || type == Type::V7) {
            normalize_ = Norm::alpha_beta(1 / 255.0f, 0.0f, ChannelType::SwapRB);
        } else if (type == Type::X) {
            // float mean[] = {0.485, 0.456, 0.406};
            // float std[]  = {0.229, 0.224, 0.225};
            // normalize_ = Norm::mean_std(mean, std, 1/255.0f, ChannelType::Invert);
            normalize_ = Norm::None();
        } else {
            INFO("Unsupport type %d", type);
        }

        auto input_dim       = trt_->static_dims(0);
        network_output_dims_ = trt_->static_dims(1);
        num_classes_         = network_output_dims_[2] - 5;
        input_width_         = input_dim[3];
        input_height_        = input_dim[2];
        return true;
    }

    virtual BoxArray forward(const Image& image, void* stream = nullptr) override {
        auto output = forwards({image}, stream);
        if (output.empty())
            return {};
        return output[0];
    }

    virtual vector<BoxArray> forwards(const vector<Image>& images, void* stream = nullptr) override {
        int batch_size = images.size();
        if (batch_size == 0)
            return {};

        auto input_dims = trt_->static_dims(0);
        input_dims[0]   = batch_size;
        trt_->set_run_dims(0, input_dims);

        adjust_memory();

        hipStream_t stream_ = (hipStream_t)stream;
        for (int i = 0; i < images.size(); ++i)
            preprocess(i, images[i], preprocess_buffers_[i], stream);

        float* network_output = predict_buffer_.gpu();
        if (!trt_->forward({input_buffer_.gpu(), network_output}, stream)) {
            INFO("Failed to tensorRT forward.");
            return {};
        }

        for (int ib = 0; ib < batch_size; ++ib) {
            float* boxarray_device       = output_boxarray_.gpu() + ib * (32 + MAX_IMAGE_BOXES * NUM_BOX_ELEMENT);
            float* boxarray_host         = output_boxarray_.cpu() + ib * (32 + MAX_IMAGE_BOXES * NUM_BOX_ELEMENT);
            float* affine_matrix_device  = (float*)preprocess_buffers_[ib]->gpu();
            float* network_output_device = network_output + ib * (network_output_dims_[1] * network_output_dims_[2]);
            checkRuntime(hipMemsetAsync(boxarray_device, 0, sizeof(int), stream_));
            decode_kernel_invoker(network_output_device, network_output_dims_[1], num_classes_, confidence_threshold_,
                                  nms_threshold_, affine_matrix_device, boxarray_device, MAX_IMAGE_BOXES, stream_);
        }
        checkRuntime(hipMemcpyAsync(output_boxarray_.cpu(), output_boxarray_.gpu(), output_boxarray_.gpu_bytes(),
                                     hipMemcpyDeviceToHost, stream_));
        checkRuntime(hipStreamSynchronize(stream_));

        vector<BoxArray> arrout(batch_size);
        for (int ib = 0; ib < batch_size; ++ib) {
            float* parray    = output_boxarray_.cpu() + ib * (32 + MAX_IMAGE_BOXES * NUM_BOX_ELEMENT);
            int count        = min(MAX_IMAGE_BOXES, (int)*parray);
            BoxArray& output = arrout[ib];
            output.reserve(count);
            for (int i = 0; i < count; ++i) {
                float* pbox  = parray + 1 + i * NUM_BOX_ELEMENT;
                int label    = pbox[5];
                int keepflag = pbox[6];
                if (keepflag == 1) {
                    output.emplace_back(pbox[0], pbox[1], pbox[2], pbox[3], pbox[4], label);
                }
            }
        }
        return arrout;
    }
};

Infer* loadraw(const std::string& engine_file, Type type, float confidence_threshold, float nms_threshold) {
    InferImpl* impl = new InferImpl();
    if (!impl->load(engine_file, type, confidence_threshold, nms_threshold)) {
        delete impl;
        impl = nullptr;
    }
    return impl;
}

shared_ptr<Infer> load(const string& engine_file, Type type, float confidence_threshold, float nms_threshold) {
    return std::shared_ptr<InferImpl>((InferImpl*)loadraw(engine_file, type, confidence_threshold, nms_threshold));
}

std::tuple<uint8_t, uint8_t, uint8_t> hsv2bgr(float h, float s, float v) {
    const int h_i = static_cast<int>(h * 6);
    const float f = h * 6 - h_i;
    const float p = v * (1 - s);
    const float q = v * (1 - f * s);
    const float t = v * (1 - (1 - f) * s);
    float r, g, b;
    switch (h_i) {
        case 0:
            r = v;
            g = t;
            b = p;
            break;
        case 1:
            r = q;
            g = v;
            b = p;
            break;
        case 2:
            r = p;
            g = v;
            b = t;
            break;
        case 3:
            r = p;
            g = q;
            b = v;
            break;
        case 4:
            r = t;
            g = p;
            b = v;
            break;
        case 5:
            r = v;
            g = p;
            b = q;
            break;
        default:
            r = 1;
            g = 1;
            b = 1;
            break;
    }
    return make_tuple(static_cast<uint8_t>(b * 255), static_cast<uint8_t>(g * 255), static_cast<uint8_t>(r * 255));
}

std::tuple<uint8_t, uint8_t, uint8_t> random_color(int id) {
    float h_plane = ((((unsigned int)id << 2) ^ 0x937151) % 100) / 100.0f;
    float s_plane = ((((unsigned int)id << 3) ^ 0x315793) % 100) / 100.0f;
    return hsv2bgr(h_plane, s_plane, 1);
}

};  // namespace yolo
